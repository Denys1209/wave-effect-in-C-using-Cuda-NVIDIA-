#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "hip/hip_runtime.h"
#include "book.h"
#include "cpu_anim.h"

#define DIM 800
#define PI 3.1415926535897932f

__global__ void kernel(unsigned char *ptr, int ticks)
{
	// block  [12, 8]
	//    ...
	//    thread [3, 5]

	// left 3t + 12b*16t = 195t
	// up   5t + 8b*16t  = 128t
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	// now calculate the value at that position
	float fx = x - DIM / 2;
	float fy = y - DIM / 2;
	float d = sqrtf(fx * fx + fy * fy);
	unsigned char grey = (unsigned char)(128.0f + 127.0f *
		cos(d / 10.0f - ticks / 7.0f) /
		(d / 10.0f + 1.0f));
	ptr[offset * 4 + 0] = grey;
	ptr[offset * 4 + 1] = grey;
	ptr[offset * 4 + 2] = grey;
	ptr[offset * 4 + 3] = 255;
}

struct DataBlock
{
	unsigned char   *dev_bitmap;
	CPUAnimBitmap  *bitmap;
};

void generate_frame(DataBlock *d, int ticks)
{
	dim3    blocks(DIM / 16, DIM / 16);
	dim3    threads(16, 16);
	kernel << <blocks, threads >> > (d->dev_bitmap, ticks);

	HANDLE_ERROR(hipMemcpy(d->bitmap->get_ptr(),
		d->dev_bitmap,
		d->bitmap->image_size(),
		hipMemcpyDeviceToHost));
}

// clean up memory allocated on the GPU
void cleanup(DataBlock *d)
{
	HANDLE_ERROR(hipFree(d->dev_bitmap));
}

int main(void)
{
	DataBlock   data;
	CPUAnimBitmap  bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;

	HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap,
		bitmap.image_size()));

	bitmap.anim_and_exit((void(*)(void*, int))generate_frame,
		(void(*)(void*))cleanup);
}